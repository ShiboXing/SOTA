#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z)
{
    return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z)
{
    const auto s = sigmoid(z);
    return (1.0 - s) * s;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh(scalar_t z)
{
    const auto t = tanh(z);
    return 1 - (t * t);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t elu(scalar_t z, scalar_t alpha = 1.0)
{
    return fmax(0.0, z) + fmin(0.0, alpha * (exp(z) - 1.0));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_elu(scalar_t z, scalar_t alpha = 1.0)
{
    const auto e = exp(z);
    const auto d_relu = z < 0.0 ? 0.0 : 1.0;
    return d_relu + (((alpha * (e - 1.0)) < 0.0) ? (alpha * e) : 0.0);
}