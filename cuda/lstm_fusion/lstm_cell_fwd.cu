#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

using namespace std;

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t &z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__global__ void lstm_cell_act_fwd(
    const scalar_t* __restrict__ gates,
    size_t state_size) {
  const int column = blockIdx.x * blockDim.x + threadIdx.x;
  const int index = blockIdx.y * state_size + column;
  const int gates_row = blockIdx.y * (state_size * 3);
  cout << blockDim.x << " " << blockIdx.x << " " << blockDim.y << " " << blockIdx.y << "\n";
  cout << threadIdx.x << " " << threadIdx.y << "\n";
  // if (column < state_size) {
  //   input_gate[index] = sigmoid(gates[gates_row + column]);
  //   output_gate[index] = sigmoid(gates[gates_row + state_size + column]);
  //   candidate_cell[index] = elu(gates[gates_row + 2 * state_size + column]);
  //   new_cell[index] =
  //       old_cell[index] + candidate_cell[index] * input_gate[index];
  //   new_h[index] = tanh(new_cell[index]) * output_gate[index];
  // }
}

vector<at::Tensor> lstm_cell_act_forward_cuda(
    torch::Tensor &gates,
    torch::Tensor &c_prev)
{
    // cout << "i_gate: " << i_gate << "\n";
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int threads = prop.maxThreadsPerBlock;
    const dim3 blocks((gates.size(0) + threads - 1) / threads, gates.size(1));

    AT_DISPATCH_FLOATING_TYPES(gates.options(), "lstm_cell_act_forward", ([&] {
        lstm_cell_act_fwd<scalar_t><<<blocks, threads>>>(
            gates.data<scalar_t>());
    }));
    
    vector<torch::Tensor> chunks = torch::chunk(gates, 4, 2);
    torch::Tensor i_gate = chunks[0], 
        f_gate = chunks[1],
        c_gate = chunks[2],
        o_gate = chunks[3];
    i_gate = at::sigmoid(i_gate);
    f_gate = at::sigmoid(f_gate);
    c_gate = at::tanh(c_gate);
    o_gate = at::sigmoid(o_gate);
    auto C = f_gate * c_prev + i_gate * c_gate; 
    auto H = o_gate * at::tanh(C);
    return {H, C};
}